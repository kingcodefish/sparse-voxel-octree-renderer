#include "hip/hip_runtime.h"
#include <cnpy.h>
#include <torch/torch.h>

#include <string>
#include <iostream>

#include <cub/device/device_scan.cuh>

using namespace std;

using point_data = ushort4;
using morton_code = unsigned long long;

/**
 * Conventions:
 * d_ = device
 * h_ = host
 */

/**
 * Does the CUDA kernels fail if the blockDim < 1024? :/
 */

const uint8_t MAX_LEVELS = 8;

static __inline__ __host__ __device__ point_data make_point_data(ushort x, ushort y, ushort z)
{
    point_data p;
    p.x = x;
    p.y = y;
    p.z = z;
    p.w = 0;

    return p;
}

static __inline__ __host__ __device__ point_data to_point(morton_code mcode)
{
    point_data p = make_point_data(0, 0, 0);

    for (int i = 0; i < 16; ++i)
    {
        p.x |= (mcode & (0x1ull << (3 * i + 2))) >> (2 * i + 2);
        p.y |= (mcode & (0x1ull << (3 * i + 1))) >> (2 * i + 1);
        p.z |= (mcode & (0x1ull << (3 * i + 0))) >> (2 * i + 0);
    }

    return p;
}

__global__ void d_scan_nodes(const uint numBytes, const uint8_t* d_octree, uint* d_info)
{
    uint tidx = blockIdx.x * 1024 + threadIdx.x;

    if (tidx < numBytes)
        d_info[tidx] = __popc(d_octree[tidx]);
}

__global__ void d_morton_to_point(const uint psize, morton_code *dataIn, point_data* dataOut)
{
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidx < psize)
        dataOut[tidx] = to_point(dataIn[tidx]);
}

__global__ void d_nodes_to_morton(const uint psize, const uint8_t* d_octreeData, const uint* d_prefixSum,
                                  const morton_code* d_mDataIn, morton_code* d_mDataOut)
{
    uint tidx = blockIdx.x * 1024 + threadIdx.x;

    if (tidx < psize)
    {
        uint8_t bits = d_octreeData[tidx];
        morton_code code = d_mDataIn[tidx];
        int addr = d_prefixSum[tidx];

        for (int i = 7; i >= 0; --i)
        {
            if (bits & (0x1 << i))
                d_mDataOut[addr--] = 8 * code + i;
        }
    }
}

struct SPC
{
    SPC() {}

    /**
     * Load NPZ file for Structured Point Cloud
     * 
     * What is the structure of this?
     * b0, b1, cc, cf, octree, pyramid, w0, w1
     * Yeah, I'm as clueless as you, but I'm hoping octree
     * is the only useful thing in this.
     * 
     * A note, the original used at::Tensor, but torch::Tensor is identical.
     * 
     * Arguments:
     * path - The path to the .npz file relative to the executable.
     */
    void load_npz(string path)
    {
        // Load in the npz file with CNPY
        cnpy::npz_t file = cnpy::npz_load(path);

        // Pull out the relevant array variables (the NPYs).
        cnpy::NpyArray octreeArray = file["octree"];
        uint8_t* octree = octreeArray.data<uint8_t>();

        // Debugging information
        cout << "Octree Data --------" << endl;
        cout << "Length: " << octreeArray.num_vals << endl;

        // Move the octree data from the CPU to the GPU
        m_octree = torch::zeros({ static_cast<long>(octreeArray.num_vals) }, torch::device(torch::kCUDA).dtype(torch::kByte));
        uint8_t* octreeDest = reinterpret_cast<uint8_t*>(m_octree.data_ptr<uint8_t>());
        hipMemcpy(octreeDest, octree, octreeArray.num_vals, hipMemcpyHostToDevice);

        vector<torch::Tensor> tmp;
        tmp = set_geometry(m_octree);
    }

    /**
     * Encode the Morton encoding of the octree to the
     * one-dimensional representation for rendering.
     */
    vector<torch::Tensor> set_geometry(torch::Tensor octree)
    {
        uint8_t* octreeData = octree.data_ptr<uint8_t>();
        m_osize = octree.size(0);

        m_info = torch::zeros({ m_osize + 1 }, torch::device(torch::kCUDA).dtype(torch::kInt32));
        m_prefixSum = torch::zeros({ m_osize + 1 }, torch::device(torch::kCUDA).dtype(torch::kInt32));
        torch::Tensor pyramidData = torch::zeros({ 2, MAX_LEVELS + 2 }, torch::device(torch::kCPU).dtype(torch::kInt32));
        
        uint* d_info = reinterpret_cast<uint*>(m_info.data_ptr<int>());
        uint* d_prefixSum = reinterpret_cast<uint*>(m_prefixSum.data_ptr<int>());
        int* h_pyramid = pyramidData.data_ptr<int>();

        void* d_temp_storage = nullptr;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_info, d_prefixSum, m_osize + 1);

        torch::Tensor temp_storage = torch::zeros({ (long)temp_storage_bytes }, torch::device(torch::kCUDA).dtype(torch::kByte));
        d_temp_storage = (void*)temp_storage.data_ptr<uint8_t>();

        // Compute exclusive sum 1 element beyond end of list to get inclusive sum starting at d_prefixSum + 1.
        // Adding 1023 before dividing the block size ensures that we use at least 1 block.
        d_scan_nodes<<<(m_osize + 1023) / 1024, 1024>>>(m_osize, octreeData, d_info);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_info, d_prefixSum, m_osize + 1);

        uint psize = 0;
        hipMemcpy(&psize, d_prefixSum + m_osize, sizeof(int), hipMemcpyDeviceToHost);
        psize++; // Plus one for root?

        torch::Tensor points = torch::zeros({ psize, 4 }, torch::device(torch::kCUDA).dtype(torch::kInt16));
        point_data* pdata = reinterpret_cast<point_data*>(points.data_ptr<short>());

        torch::Tensor mortons = torch::zeros({ psize }, torch::device(torch::kCUDA).dtype(torch::kInt64));
        morton_code* mdata = reinterpret_cast<morton_code*>(mortons.data_ptr<long>());

        int* pyramid = h_pyramid;
        int* pyramidSum = h_pyramid + MAX_LEVELS + 2;

        uint* S = d_prefixSum + 1; // This shouldn't matter?
        morton_code* M = mdata;
        uint8_t* O = octreeData;

        morton_code m0 = 0;
        hipMemcpy(M, &m0, sizeof(morton_code), hipMemcpyHostToDevice);

        int lsize = 1;
        uint currSum, prevSum = 0;

        uint sum = pyramid[0] = lsize;
        pyramidSum[0] = 0;
        pyramidSum[1] = sum;

        int level = 0;
        while (sum <= m_osize)
        {
            d_nodes_to_morton<<<(lsize + 1023) / 1024, 1024>>>(lsize, O, S, M, mdata);
            O += lsize;
            S += lsize;
            M += lsize;

            hipMemcpy(&currSum, d_prefixSum + prevSum + 1, sizeof(int), hipMemcpyDeviceToHost);

            lsize = currSum - prevSum;
            prevSum = currSum;

            pyramid[++level] = lsize;
            sum += lsize;
            pyramidSum[level + 1] = sum;
        }

        uint totalPoints = pyramidSum[level + 1];

        d_morton_to_point<<<(totalPoints + 1023) / 1024, 1024>>>(totalPoints, mdata, pdata);
        hipGetLastError();

        // Assemble output tensors
        std::vector<torch::Tensor> result;
        result.push_back(points);
        result.push_back(pyramidData.index({
            torch::indexing::Slice(torch::indexing::None),
            torch::indexing::Slice(torch::indexing::None,
            level + 2)
        }).contiguous());

        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }

        return result;
    }

private:
    torch::Tensor m_octree;
    torch::Tensor m_points;
    torch::Tensor m_info;
    torch::Tensor m_prefixSum;
    torch::Tensor m_pyramid;

    uint8_t       m_level = 0;
    uint8_t       m_psize = 0;
    uint8_t       m_osize = 0;
};

int main()
{
    SPC* spc = new SPC();
    spc->load_npz("armadillo.npz");
}